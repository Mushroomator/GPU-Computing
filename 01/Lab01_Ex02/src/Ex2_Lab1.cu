#include "hip/hip_runtime.h"
// You need to write a simple program to perform computation with 1D array in CPU and GPU, then compare the result.

// includes, system
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

// Part 3 of 5: implement the kernel
__global__ void calculate1DKernel(int *d_a)
{
  	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	d_a[idx] = 1000 * blockIdx.x + threadIdx.x;
	printf("%u:  \t %u = 1000 * %u + %u\n", idx, d_a[idx] , blockIdx.x, threadIdx.x);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    // pointer for host memory
    int *h_a;

    // pointer for device memory
    int *d_a;

    // define grid and block size
    int numBlocks = 8;
    int numThreadsPerBlock = 8;

    // Part 1 of 5: allocate host and device memory
    size_t size = numBlocks * numThreadsPerBlock * sizeof(int);
    hipHostMalloc((void **)&h_a, size);
    hipMalloc((void **)&d_a, size);


    // Part 2 of 5: launch kernel
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    calculate1DKernel<<<numBlocks, numThreadsPerBlock>>>(d_a);    

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    checkCUDAError("kernel execution");

    // Part 4 of 5: device to host copy
    hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");

    // Part 5 of 5: verify the data returned to the host is correct
    // i represents blockIdx.x
    for(int i = 0; i < numBlocks; i++){
	// j represents threadIdx.x    
    	for(int j = 0; j < numThreadsPerBlock; j++){
		int idx = i * numThreadsPerBlock + j;
		printf("%u\n", idx);
		assert(h_a[idx] == (1000 * i + j));
	}	
    }

    // free device memory
    hipFree(d_a);

    // free host memory
    hipHostFree(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}
