
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h> 

#define SIZE 10

// Convert and mod
__global__ void add_kernel(uint32_t *d_c, uint32_t *d_a, uint32_t *d_b) {   
   uint32_t tid = threadIdx.x;

   d_c[tid] = d_a[tid] + d_b[tid];
}

int main(int argc, char* argv[]) {
    int i;
    uint32_t *h_a, *h_b, *h_c;  // host pointer
    uint32_t *d_a, *d_b, *d_c;  // device pointer

    hipHostMalloc((void**)&h_a, SIZE * sizeof(uint32_t), hipHostMallocDefault);   
    hipHostMalloc((void**)&h_b, SIZE * sizeof(uint32_t), hipHostMallocDefault);   
    hipHostMalloc((void**)&h_c, SIZE * sizeof(uint32_t), hipHostMallocDefault);   
    hipMalloc((void**)&d_a, SIZE * sizeof(uint32_t));   
    hipMalloc((void**)&d_b, SIZE * sizeof(uint32_t));   
    hipMalloc((void**)&d_c, SIZE * sizeof(uint32_t));   

    for(i=0; i<SIZE; i++) 
    {
      h_a[i] = i; 
      h_b[i] = i;
    }

    hipMemcpy(d_a, h_a, SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);  
    hipMemcpy(d_b, h_b, SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);     

    add_kernel<<<1, SIZE>>>(d_c, d_a, d_b);
    hipMemcpy(h_c, d_c, SIZE * sizeof(uint32_t), hipMemcpyDeviceToHost);  

    printf("Results:\n");
    for(i=0; i<SIZE; i++) printf("%u ", h_c[i]);

    hipFree(h_a);    hipFree(h_b);        hipFree(h_c);
    hipFree(d_a);    hipFree(d_b);        hipFree(d_c);    
    hipDeviceReset();
    return 0;
}

