#include "hip/hip_runtime.h"
// You need to write a simple program to perform computation with 1D array in CPU and GPU, then compare the result.

// includes, system
#include <stdio.h>
#include <assert.h>

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

// Part 3 of 5: implement the kernel
__global__ void calculate1DKernel(int *d_a)
{

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    // pointer for host memory
    int *h_a;

    // pointer for device memory
    int *d_a;

    // define grid and block size
    int numBlocks = 8;
    int numThreadsPerBlock = 8;

    // Part 1 of 5: allocate host and device memory


    // Part 2 of 5: launch kernel


    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    checkCUDAError("kernel execution");

    // Part 4 of 5: device to host copy
	

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");

    // Part 5 of 5: verify the data returned to the host is correct

    // free device memory
    hipFree(d_a);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}
