#include <stdio.h>
#include <stdint.h>
#include <stdlib.h> 

// CUDA runtime
#include <hip/hip_runtime.h>



#define SIZE 100000000
#define THREADS_PER_BLOCK 1024

// Convert and mod
__global__ void add_kernel(uint32_t *d_c, uint32_t *d_a, uint32_t *d_b) {   
//  compute index = thread index in a block + block index * number of threads per block
	uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	// if SIZE mod THREADS PER BLOCK != 0 --> some index will not be available as one more block is used
	if(idx <= SIZE) d_c[idx] = d_a[idx] + d_b[idx];
}

int main(int argc, char* argv[]) {
    int i;
    uint32_t *h_a, *h_b, *h_c;  // host pointer
    uint32_t *d_a, *d_b, *d_c;  // device pointer

    hipHostMalloc((void**)&h_a, SIZE * sizeof(uint32_t), hipHostMallocDefault);   
    hipHostMalloc((void**)&h_b, SIZE * sizeof(uint32_t), hipHostMallocDefault);   
    hipHostMalloc((void**)&h_c, SIZE * sizeof(uint32_t), hipHostMallocDefault);   
    hipMalloc((void**)&d_a, SIZE * sizeof(uint32_t));   
    hipMalloc((void**)&d_b, SIZE * sizeof(uint32_t));   
    hipMalloc((void**)&d_c, SIZE * sizeof(uint32_t));   

    for(i=0; i<SIZE; i++) 
    {
      h_a[i] = i; 
      h_b[i] = i;
    }

    hipMemcpy(d_a, h_a, SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);  
    hipMemcpy(d_b, h_b, SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);     

//  <<< [Number of blocks], [Number of threads per block] >>>
    add_kernel<<<(SIZE / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(d_c, d_a, d_b);
    hipMemcpy(h_c, d_c, SIZE * sizeof(uint32_t), hipMemcpyDeviceToHost);  

    printf("\n----------\nResults CPU:\n");
    for(i=0; i<SIZE; i++) printf("%u: %u ",i , h_c[i]);

    hipFree(h_a);    hipFree(h_b);        hipFree(h_c);
    hipFree(d_a);    hipFree(d_b);        hipFree(d_c);    
    hipDeviceReset();
    return 0;
}

